#include <iostream>
#include <string>
#include <vector>
#include <fstream>
#include <sstream>
#include <cstring>
#include <cmath>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>



using namespace std;

#define gridmax 32768

__global__ void check_redun(uint num_match, uint* segments, uint* timeoday, uint* days, float* values, uint* num_redun){
    uint i = blockIdx.x + threadIdx.x;
    if(i < num_match){
        for(int j = i + 1; j < num_match; ++j){
            if((segments[i] == segments[j]) && (timeoday[i] == timeoday[j]) && (days[i] == days[j])){
                segments[j] = 10000;
                atomicAdd(&(values[j]), values[i]);
                atomicAdd(&(num_redun[0]), 1);
            }
        }
    }
    printf("%u", num_redun[0]);

}

int main(int argc, char **argv){
    
    // Read gps file and load the data into lists
    ifstream inFile_gps(argv[1], ios::in);
    string lineStr;
    vector<string> item;
    uint num_gps = 0;

    getline(inFile_gps, lineStr);
    while(getline(inFile_gps, lineStr)){
        item.push_back(lineStr);
        num_gps++;
    }
    cout<<"The number of gps is: "<<num_gps<<endl;

    // 创建数组存储轨迹数据的各列
    double *Data_lng = nullptr;
    Data_lng = (double*)malloc(sizeof(double) * num_gps);
    memset(Data_lng, 0, sizeof(double) * num_gps);

    double *Data_lat = nullptr;
    Data_lat = (double*)malloc(sizeof(double) * num_gps);
    memset(Data_lat, 0, sizeof(double) * num_gps);

    uint *Data_time = nullptr;
    Data_time = (uint*)malloc(sizeof(uint) * num_gps);
    memset(Data_time, 0, sizeof(uint) * num_gps);

    int *Data_day = nullptr;
    Data_day = (int*)malloc(sizeof(int) * num_gps);
    memset(Data_day, 0, sizeof(int) * num_gps);

    // 存储轨迹数据
    uint round = 0;
    for (auto it = item.begin(); it != item.end(); it++){
        string str;
        istringstream istr(*it);
        getline(istr, str, ',');
        // cout << str << ' ';
        Data_lng[round] = atof(str.c_str());

        getline(istr, str, ',');
        // cout << str << ' ';
        Data_lat[round] = atof(str.c_str());

        getline(istr, str, ',');
        // cout << str <<endl;
        Data_time[round] = atoi(str.c_str());

        getline(istr, str, ',');
        Data_day[round] = atoi(str.c_str());
        round++;
    }

    // 输出每个数组的前五个元素.检查是否正确
    for(uint i = 0; i < 5; i++){
        cout<<Data_lng[i]<<' ';
    }
    cout<<endl;
    for(uint i = 0; i < 5; i++){
        cout<<Data_lat[i]<<' ';
    }
    cout<<endl;
    for(uint i = 0; i < 5; i++){
        cout<<Data_time[i]<<' ';
    }
    cout<<endl;
    for(uint i = 0; i < 5; i++){
        cout<<Data_day[i]<<' ';
    }
    cout<<endl;

    cout<<"finish read the gps file"<<endl;

    // 创建数组存储张量的各列元素
    uint *segments = nullptr;
    segments = (uint*)malloc(sizeof(uint) * num_gps);
    memset(segments, 10000, sizeof(uint) * num_gps);
    uint *timeoday = nullptr;
    timeoday = (uint*)malloc(sizeof(uint) * num_gps);
    memset(timeoday, 0, sizeof(uint) * num_gps);
    uint *days = nullptr;
    days = (uint*)malloc(sizeof(uint) * num_gps);
    memset(days, 0, sizeof(uint) * num_gps);
    float *values = nullptr;
    values = (float*)malloc(sizeof(float) * num_gps);
    memset(values, 0, sizeof(float) * num_gps);




    cout<<"create arrays to store tensor values and index"<<endl;

    // Read link file and load the data into lists
    ifstream inFile_road(argv[2], ios::in);
    string lineStr_road;
    vector<string> item_road;
    uint num_road = 0;

    // 存取道路文件信息
    getline(inFile_road, lineStr_road);
    while(getline(inFile_road, lineStr_road)){
        item_road.push_back(lineStr_road);
        num_road++;
    }
    cout<<"The number of road is: "<<num_road<<endl;

    // 创建数组存储道路信息
    double *Road_lng = nullptr;
    Road_lng = (double*)malloc(sizeof(double) * num_road);
    memset(Road_lng, 0, sizeof(double) * num_road);

    double *Road_lat = nullptr;
    Road_lat = (double*)malloc(sizeof(double) * num_road);
    memset(Road_lat, 0, sizeof(double) * num_road);

    uint *Road_id = nullptr;
    Road_id = (uint*)malloc(sizeof(uint) * num_road);
    memset(Road_id, 0, sizeof(uint) * num_road);

    // 存储道路信息
    uint round_r = 0;
    for (auto it_r = item_road.begin(); it_r != item_road.end(); it_r++){
        string str_r;
        istringstream istr_r(*it_r);
        getline(istr_r, str_r, ',');
        Road_id[round_r] = atoi(str_r.c_str());

        getline(istr_r, str_r, ',');
        Road_lng[round_r] = atof(str_r.c_str());

        getline(istr_r, str_r, ',');
        Road_lat[round_r] = atof(str_r.c_str());

        round_r++;
    }

    // 输出数组前五个元素验证是否正确
    for(int i = 0; i < 5; ++i)
        cout<<Road_lng[i]<<' ';
    cout<<endl;
    for(int i = 0; i < 5; ++i)
        cout<<Road_lat[i]<<' ';
    cout<<endl;
    for(int i = 0; i < 5; ++i)
        cout<<Road_id[i]<<' ';
    cout<<endl;
    cout<<"The number of road is: "<<round_r<<endl;
    cout<<"finish load the road data\n";

    // 判断segments数组赋值是否有错误
    // for(uint i = 0; i < num_gps; ++i){
    //     if(segments[i] != 10000){
    //         cout<<"segments value error\n";
    //         return 0;
    //     }
    // }
    // for(int i = 0; i < num_road; ++i){
    //     cout<<Road_id[i]<<' ';
    // }
    //进行路网匹配
    uint num_match = 0;
    uint i, j;
    // #pragma omp parallel for num_threads(16), private(i,j)
    for(i = 0; i < num_gps; ++i){
        for(j = 0; j < num_road; ++j){
            if((fabs(Data_lng[i] - Road_lng[j]) <= 0.005) && (fabs(Data_lat[i] - Road_lat[j]) <= 0.005)){
                segments[num_match] = Road_id[j];
                // cout<<segments[num_match]<<' '<<Road_id[j]<<endl;
                timeoday[num_match] = Data_time[i];
                days[num_match] = Data_day[i];
                values[num_match] = 1;
                num_match++;
                break;
            }
        }
    }

    cout<<"num_match = "<<num_match<<endl;
    
    // uint num_repetition = 0;
    uint re_num[1] = {0};
    // #pragma omp parallel for num_threads(16), private(i)
    // for(i = 0; i < num_match; ++i){
    //     for(j = i + 1; j < num_match; ++j){
    //         if((segments[i] == segments[j]) && (timeoday[i] == timeoday[j]) && (days[i] == days[j])){
    //             segments[j] = 10000;
    //             values[i]++;
    //             num_repetition++;
    //         }
    //     }
    // }
    
    uint *segments_d;
    hipMalloc((void**)&segments_d, sizeof(uint) * num_gps);
    hipMemcpy(segments_d, segments, sizeof(uint) * num_gps, hipMemcpyHostToDevice);
    uint *timeoday_d;
    hipMalloc((void**)&timeoday_d, sizeof(uint) * num_gps);
    hipMemcpy(timeoday_d, timeoday, sizeof(uint) * num_gps, hipMemcpyHostToDevice);
    uint *days_d;
    hipMalloc((void**)&days_d, sizeof(uint) * num_gps);
    hipMemcpy(days_d, days, sizeof(uint) * num_gps, hipMemcpyHostToDevice);
    float *values_d;
    hipMalloc((void**)&values_d, sizeof(float) * num_gps);
    hipMemcpy(values_d, values, sizeof(float) * num_gps, hipMemcpyHostToDevice);
    uint *re_num_d;
    hipMalloc((void**)&re_num_d, sizeof(uint) * 1);
    hipMemcpy(re_num_d, re_num, sizeof(uint) * 1, hipMemcpyHostToDevice);
    

    // uint *segments_r = nullptr;
    // segments_r = (uint*)malloc(sizeof(uint) * num_gps);
    // memset(segments_r, 10000, sizeof(uint) * num_gps);
    // uint *timeoday_r = nullptr;
    // timeoday_r = (uint*)malloc(sizeof(uint) * num_gps);
    // memset(timeoday_r, 0, sizeof(uint) * num_gps);
    // uint *days_r = nullptr;
    // days_r = (uint*)malloc(sizeof(uint) * num_gps);
    // memset(days_r, 0, sizeof(uint) * num_gps);
    // float *values_r = nullptr;
    // values_r = (float*)malloc(sizeof(float) * num_gps);
    // memset(values_r, 0, sizeof(float) * num_gps);
    
    cout<<"data memcpy to device"<<endl;
    dim3 block(256);
    dim3 grid;
    if(num_match < gridmax)
        grid.x = num_match;
    else
        grid.x = gridmax - 10000;
    check_redun<<<grid, block>>>(num_match, segments_d, timeoday_d, days_d, values_d, re_num_d);

    cout<<"finish gpu"<<endl;

    hipMemcpy(segments, segments_d, sizeof(uint) * num_gps, hipMemcpyDeviceToHost);
    hipMemcpy(timeoday, timeoday_d, sizeof(uint) * num_gps, hipMemcpyDeviceToHost);
    hipMemcpy(days, days_d, sizeof(uint) * num_gps, hipMemcpyDeviceToHost);
    hipMemcpy(values, values_d, sizeof(float) * num_gps, hipMemcpyDeviceToHost);
    hipMemcpy(re_num, re_num_d, sizeof(uint) * 1, hipMemcpyDeviceToHost);


    cout<<"The number of repetition is :"<<re_num[0]<<endl;
    uint num_day = 28;
    uint num_time = 18 * 60 * 60 / 60;

    // for(uint s = 0; s < 100; ++s)
    //     cout<<segments[s]<<' ';
    // cout<<endl;

    // make the begin index from 0 to 1
    for(uint i = 0; i < num_match; ++i){
        segments[i]++;
        timeoday[i] = (timeoday[i] / 60) - 359;
        days[i]++;
    }
    
    // write into file 
    FILE *fp = fopen(argv[3], "w");
    fprintf(fp, "3\n");
    fprintf(fp, "%u %u %u\n", round_r, num_time, num_day);
    for(uint i = 0; i < num_match; ++i){
        if(segments[i] != 10001){
            fprintf(fp, "%u %u %u %.4f\n", segments[i], timeoday[i], days[i], values[i]);
        }
    }
    cout<<"finish write\n";

    free(Data_lng);
    free(Data_lat);
    free(Data_time);
    free(Data_day);
    free(Road_id);
    free(Road_lng);
    free(Road_lat);
    free(segments);
    free(timeoday);
    free(days);
    free(values);
    hipFree(segments_d);
    hipFree(timeoday_d);
    hipFree(days_d);
    hipFree(values_d);
    hipFree(re_num_d);

    return 0;
}